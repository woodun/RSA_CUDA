#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.cu"
#include <time.h>
#include "cuda_mpz.h"


long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	}
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}

	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;

	return time_interval_s + time_interval_ns;
}

//L1 enabled. (-keep -Xptxas -dlcm=ca --optimize 0)
int main (int argc, char *argv[]) {

	printf("%d", sizeof(mp_limb_t));
//	mpz_t integ;
//	mpz_init (integ);
//	mpz_clear (integ);
//
//	cuda_mpz_t test;
//	cuda_mpz_init(&test);


//	struct timespec ts1;
//	clock_gettime(CLOCK_REALTIME, &ts1);
//
//	///////input control
//	if (argc < 5){
//		exit(EXIT_FAILURE);
//	}
//
//	long x = strtol(argv[5], NULL, 10);
//	long long unsigned pairs = x;
//	unsigned thread_num = 2;
//	long long unsigned data_num = pairs * thread_num;
//
//	///////host memory
//	long long int *clockTable_h;
//	clockTable_h = (long long int*) malloc( pairs * sizeof(long long int));
//
//	cuda_mpz_t h_n;
//	cuda_mpz_t h_n_;
//	cuda_mpz_t h_r2;
//	int rl = 70;
//
//	cuda_mpz_init(&h_n);
//	cuda_mpz_init(&h_n_);
//	cuda_mpz_init(&h_r2);
//
//	///////get n
//	char n_input[] = "00000038f6e8cfba55dd0e47";
//	cuda_mpz_set_str_host(&h_n, n_input);
//
//	///////get n_
//	char n__input[] = "0000002e8457440e0d93c489";
//	cuda_mpz_set_str_host(&h_n_, n__input);
//
//	///////get r2
//	char r2_input[] = "0000003709d17d8f8686609f";
//	cuda_mpz_set_str_host(&h_r2, r2_input);
//
//	///////get e
//	char e_input[] = "101";
//	int e_bitsLength = (int)strlen(e_input);
//	int* eBits = (int *) malloc(sizeof(int) * e_bitsLength);
//
//	int* eBits_d;
//	hipMalloc((void **) &eBits_d, sizeof(int) * e_bitsLength);
//
//	int e_iterator = e_bitsLength - 1;
//	while ( e_iterator > 0){
//        if( e_input[e_bitsLength - 1 - e_iterator] == '1'){
//            eBits[e_iterator] = 1;
//        }
//        else{
//            eBits[e_iterator] = 0;
//        }
//        e_iterator--;
//	}
//	eBits[e_iterator] = 1;
//	hipMemcpy(eBits_d, eBits, sizeof(int) * e_bitsLength, hipMemcpyHostToDevice);
//
//	///////get d
//	char d_input[] = "1011011001001001010011110110010101010111001010110101111000111100001";
//
//	int d_bitsLength = (int)strlen(d_input);
//
//	int* dBits = (int *) malloc(sizeof(int) * d_bitsLength);
//
//	int* dBits_d;
//	hipMalloc((void **) &dBits_d, sizeof(int) * d_bitsLength);
//
//	int d_iterator = d_bitsLength - 1;
//	while ( d_iterator > 0){
//        if( d_input[d_bitsLength - 1 - d_iterator] == '1'){
//            dBits[d_iterator] = 1;
//        }
//        else{
//            dBits[d_iterator] = 0;
//        }
//        d_iterator--;
//	}
//	dBits[d_iterator] = 1;
//	hipMemcpy(dBits_d, dBits, sizeof(int) * d_bitsLength, hipMemcpyHostToDevice);
//
//	///////device memory
//	unsigned varSize = sizeof(cuda_mpz_t) * thread_num;
//
//	long long int *clockTable_d;
//	cuda_mpz_t *tmp;
//	cuda_mpz_t *tmp2;
//	cuda_mpz_t *d_t;
//	cuda_mpz_t *_x1_cuda_mpz;
//	cuda_mpz_t *_x2_cuda_mpz;
//	hipMalloc((void **) &clockTable_d, pairs * sizeof(long long int));
//	hipMalloc((void **) &tmp, varSize);
//	hipMalloc((void **) &tmp2, varSize);
//	hipMalloc((void **) &d_t, varSize);
//	hipMalloc((void **) &_x2_cuda_mpz, varSize);
//	hipMalloc((void **) &_x1_cuda_mpz, varSize);
//
//	init<<<1, thread_num>>>(_x1_cuda_mpz, _x2_cuda_mpz, tmp, tmp2, d_t);
//	hipDeviceSynchronize();
//
//	///////get Messages
//	long long unsigned mesSize = sizeof(cuda_mpz_t) * data_num;
//	cuda_mpz_t *myMes1_h;
//	myMes1_h = (cuda_mpz_t*) malloc (mesSize);//CPU
//
//	for(long long unsigned i = 0; i < data_num; i++){
//		cuda_mpz_init(&myMes1_h[i]);
//	}
//
//	cuda_mpz_t *myMes1_d;
//	hipMalloc((cuda_mpz_t **) &myMes1_d, mesSize);//GPU
//
//	///////get Message pairs
//	char* line = NULL;
//	size_t len = 0;
//
//	for(int i = 2; i < 5; i++){
//		FILE* fp = fopen(argv[i], "r");//input from pair storage
//		if (fp == NULL){
//			exit(EXIT_FAILURE);
//		}
//
//		long long unsigned line_num = 0;
//		while ((getline(&line, &len, fp)) != -1) {
//			line[strcspn(line, "\n")] = 0;
//			cuda_mpz_set_str_host(&myMes1_h[line_num], line);
//			line_num++;
//			if(line_num == data_num){
//				break;
//			}
//		}
//		fclose(fp);
//
//		hipMemcpy(myMes1_d, myMes1_h, mesSize, hipMemcpyHostToDevice);
//
//		MontSQMLadder<<<1, thread_num>>>(myMes1_d, pairs, _x1_cuda_mpz, _x2_cuda_mpz, tmp, tmp2, rl, h_r2, h_n, h_n_, dBits_d, d_bitsLength, clockTable_d, d_t);/////////////////////////////////////////kernel
//		hipDeviceSynchronize();
//
//		hipMemcpy(clockTable_h, clockTable_d, pairs * sizeof(long long int), hipMemcpyDeviceToHost);
//
//		long long unsigned sum = 0;
//
//		for (long long unsigned q = 0; q < pairs; q++){
//			sum += clockTable_h[q];
//		}
//
//		sum = sum / pairs;
//		printf("%llu ", sum);
//	}
//
//	////////free device
//	hipFree(clockTable_d);
//	hipFree(eBits_d);
//	hipFree(dBits_d);
//	hipFree(myMes1_d);
//	hipFree(tmp);
//	hipFree(tmp2);
//	hipFree(d_t);
//	hipFree(_x1_cuda_mpz);
//	hipFree(_x2_cuda_mpz);
//
//	////////free host
//	if (line){
//		free(line);
//	}
//	free(clockTable_h);
//	free(myMes1_h);
//	free(eBits);
//	free(dBits);
//
//	/////////////////////////////////time
//	struct timespec ts2;
//	clock_gettime(CLOCK_REALTIME, &ts2);
//
//	printf("%llu ", time_diff(ts1, ts2));

    return 0;
}

