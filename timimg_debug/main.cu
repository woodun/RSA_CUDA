#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.cu"
#include <time.h>
#include "cuda_mpz.h"
#include <gmp.h>


//nvprof --print-gpu-trace --log-file prof1.txt ./main 1000 1 > nvprof.txt
int CheckREDC(cuda_mpz_t* N, cuda_mpz_t* N_, cuda_mpz_t* T, cuda_mpz_t* tmp, cuda_mpz_t* t){

	//m = ((T & R) * N_) & R
	cuda_mpz_bitwise_truncate(t, T, RL);
	cuda_mpz_mult(tmp, N_, t);
	cuda_mpz_bitwise_truncate_eq(tmp, RL);

	//t = (T + m*N) >> L
	cuda_mpz_mult(t, tmp , N);
	cuda_mpz_add(tmp, T, t);
	cuda_mpz_bitwise_rshift(t, tmp, RL);

	if (cuda_mpz_gte(t , N)){
		return 1;
    }
	else{
	    return 0;
	}
}

int CheckDivExp(cuda_mpz_t * mes1, cuda_mpz_t * mes2, int* eBits, int eLength, cuda_mpz_t* _x1_1, cuda_mpz_t* _x1_2, cuda_mpz_t* _x2_1, cuda_mpz_t* _x2_2,
		cuda_mpz_t* _x1_1_temp, cuda_mpz_t* _x1_2_temp, cuda_mpz_t* _x2_1_temp, cuda_mpz_t* _x2_2_temp,
		cuda_mpz_t* tmp_1, cuda_mpz_t* tmp_2, cuda_mpz_t* tmp2_1, cuda_mpz_t* tmp2_2, cuda_mpz_t* r2, cuda_mpz_t* n, cuda_mpz_t* n_,  cuda_mpz_t* t_1, cuda_mpz_t* t_2, long check_pre){

	int div_count = 0;

	//mes1 * r2
	cuda_mpz_mult(tmp2_1, mes1, r2);
	//mes2 * r2
	cuda_mpz_mult(tmp2_2, mes2, r2);

	//s1_1 = CheckREDC(rmod, n, n_, mes1 * r2, l)
	//s1_2 = CheckREDC(rmod, n, n_, mes2 * r2, l)
	int s1_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);
	int s1_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);

	if (s1_1 != s1_2){
		div_count++;
	}

	//_x1_1 = REDC(rmod, n, n_, mes1 * r2, l)
	//_x1_2 = REDC(rmod, n, n_, mes2 * r2, l)
	cuda_mpz_set( _x1_1, REDC( n, n_, tmp2_1, tmp_1, t_1) );
	cuda_mpz_set( _x1_2, REDC( n, n_, tmp2_2, tmp_2, t_2) );

	//_x2_1 = _x1_1 * _x1_1
	//_x2_2 = _x1_2 * _x1_2
	cuda_mpz_mult(tmp2_1, _x1_1, t_1);
	cuda_mpz_mult(tmp2_2, _x1_2, t_2);

	//s2_1 = CheckREDC(rmod, n, n_, _x2_1, l)
	//s2_2 = CheckREDC(rmod, n, n_, _x2_2, l)
	int s2_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);
	int s2_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);

	if (s2_1 != s2_2){
		div_count++;
	}

	//_x2_1 = REDC(rmod, n, n_, _x2_1, l)
	//_x2_2 = REDC(rmod, n, n_, _x2_2, l)
	cuda_mpz_set( _x2_1, REDC( n, n_, tmp2_1, tmp_1, t_1) );
	cuda_mpz_set( _x2_2, REDC( n, n_, tmp2_2, tmp_2, t_2) );

	//for i in e_b[1:]:
	for(int i = 1; i < eLength; i++){ //big endian

		if(eBits[i] == 0){
			//_x2_1 = _x1_1 * _x2_1
			//_x2_2 = _x1_2 * _x2_2
			cuda_mpz_mult(tmp2_1, _x1_1, _x2_1);
			cuda_mpz_mult(tmp2_2, _x1_2, _x2_2);

			//s2_1 = CheckREDC(rmod, n, n_, _x2_1, l)
			//s2_2 = CheckREDC(rmod, n, n_, _x2_2, l)
			s2_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);
			s2_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);

			if (s2_1 != s2_2){
//				return 0;
				div_count++;
			}

			//_x2_1 = REDC(rmod, n, n_, _x2_1, l)
			//_x2_2 = REDC(rmod, n, n_, _x2_2, l)
			cuda_mpz_set( _x2_1, REDC( n, n_, tmp2_1, tmp_1, t_1) );
			cuda_mpz_set( _x2_2, REDC( n, n_, tmp2_2, tmp_2, t_2) );

			//_x1_1 = _x1_1 * _x1_1
			//_x1_2 = _x1_2 * _x1_2
			cuda_mpz_set( tmp_1, _x1_1);
			cuda_mpz_mult(tmp2_1, _x1_1, tmp_1);
			cuda_mpz_set( tmp_2, _x1_2);
			cuda_mpz_mult(tmp2_2, _x1_2, tmp_2);

			//s1_1 = CheckREDC(rmod, n, n_, _x1_1, l)
			//s1_2 = CheckREDC(rmod, n, n_, _x1_2, l)
			s1_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);
			s1_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);

			if (s1_1 != s1_2){
//				return 0;
				div_count++;
			}

			//_x1_1 = REDC(rmod, n, n_, _x1_1, l)
			//_x1_2 = REDC(rmod, n, n_, _x1_2, l)
			cuda_mpz_set( _x1_1, REDC( n, n_, tmp2_1, tmp_1, t_1) );
			cuda_mpz_set( _x1_2, REDC( n, n_, tmp2_2, tmp_2, t_2) );
		} else{
			//_x1_1 = _x1_1 * _x2_1
			//_x1_2 = _x1_2 * _x2_2
			cuda_mpz_mult(tmp2_1, _x1_1, _x2_1);
			cuda_mpz_mult(tmp2_2, _x1_2, _x2_2);

			//s1_1 = CheckREDC(rmod, n, n_, _x1_1, l)
			//s1_2 = CheckREDC(rmod, n, n_, _x1_2, l)
			s1_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);
			s1_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);

			if (s1_1 != s1_2){
//				return 0;
				div_count++;
			}

			//_x1_1 = REDC(rmod, n, n_, _x1_1, l)
			//_x1_2 = REDC(rmod, n, n_, _x1_2, l)
			cuda_mpz_set( _x1_1, REDC( n, n_, tmp2_1, tmp_1, t_1) );
			cuda_mpz_set( _x1_2, REDC( n, n_, tmp2_2, tmp_2, t_2) );

			//_x2_1 = _x2_1 * _x2_1
			//_x2_2 = _x2_2 * _x2_2
			cuda_mpz_set( tmp_1, _x2_1);
			cuda_mpz_mult(tmp2_1, _x2_1, tmp_1);
			cuda_mpz_set( tmp_2, _x2_2);
			cuda_mpz_mult(tmp2_2, _x2_2, tmp_2);


			//s2_1 = CheckREDC(rmod, n, n_, _x2_1, l)
			//s2_2 = CheckREDC(rmod, n, n_, _x2_2, l)
			s2_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);
			s2_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);

			if (s2_1 != s2_2){
//				return 0;
				div_count++;
			}

			//_x2_1 = REDC(rmod, n, n_, _x2_1, l)
			//_x2_2 = REDC(rmod, n, n_, _x2_2, l)
			cuda_mpz_set( _x2_1, REDC( n, n_, tmp2_1, tmp_1, t_1) );
			cuda_mpz_set( _x2_2, REDC( n, n_, tmp2_2, tmp_2, t_2) );
		}
	}

	if(div_count != eLength && check_pre == 1){ //total divergence number
		return 0;
	}

	//_x1_1_temp = _x1_1
	cuda_mpz_set( _x1_1_temp, _x1_1);
	//_x2_1_temp = _x2_1
	cuda_mpz_set( _x2_1_temp, _x2_1);
	//_x1_2_temp = _x1_2
	cuda_mpz_set( _x1_2_temp, _x1_2);
	//_x2_2_temp = _x2_2
	cuda_mpz_set( _x2_2_temp, _x2_2);

	//simulate exp bit 0
	//_x2_1 = _x1_1 * _x2_1
	cuda_mpz_mult(tmp2_1, _x1_1, _x2_1);
	//d0_s2_1 = CheckREDC(rmod, n, n_, _x2_1, l)
	int d0_s2_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);
	//_x1_1 = _x1_1 * _x1_1
	cuda_mpz_set( tmp_1, _x1_1);
	cuda_mpz_mult(tmp2_1, _x1_1, tmp_1);
	//d0_s1_1 = CheckREDC(rmod, n, n_, _x1_1 ,l)
	int d0_s1_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);

	//_x2_2 = _x1_2 * _x2_2
	cuda_mpz_mult(tmp2_2, _x1_2, _x2_2);
	//d0_s2_2 = CheckREDC(rmod, n, n_, _x2_2, l)
	int d0_s2_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);
	//_x1_2 = _x1_2 * _x1_2
	cuda_mpz_set( tmp_2, _x1_2);
	cuda_mpz_mult(tmp2_2, _x1_2, tmp_2);
	//d0_s1_2 = CheckREDC(rmod, n, n_, _x1_2 ,l)
	int d0_s1_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);

	//simulate exp bit 1
	//_x1_1 = _x1_1_temp
	cuda_mpz_set( _x1_1, _x1_1_temp);
	//_x2_1 = _x2_1_temp
	cuda_mpz_set( _x2_1, _x2_1_temp);
	//_x1_2 = _x1_2_temp
	cuda_mpz_set( _x1_2, _x1_2_temp);
	//_x2_2 = _x2_2_temp
	cuda_mpz_set( _x2_2, _x2_2_temp);

	//_x1_1 = _x1_1 * _x2_1
	cuda_mpz_mult(tmp2_1, _x1_1, _x2_1);
	//d1_s1_1 = CheckREDC(rmod, n, n_, _x1_1, l)
	int d1_s1_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);
	//_x2_1 = _x2_1 * _x2_1
	cuda_mpz_set( tmp_1, _x2_1);
	cuda_mpz_mult(tmp2_1, _x2_1, tmp_1);
	//d1_s2_1 = CheckREDC(rmod, n, n_, _x2_1, l)
	int d1_s2_1 = CheckREDC( n, n_, tmp2_1, tmp_1, t_1);

	//_x1_2 = _x1_2 * _x2_2
	cuda_mpz_mult(tmp2_2, _x1_2, _x2_2);
	//d1_s1_2 = CheckREDC(rmod, n, n_, _x1_2, l)
	int d1_s1_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);
	//_x2_2 = _x2_2 * _x2_2
	cuda_mpz_set( tmp_2, _x2_2);
	cuda_mpz_mult(tmp2_2, _x2_2, tmp_2);
	//d1_s2_2 = CheckREDC(rmod, n, n_, _x2_2, l)
	int d1_s2_2 = CheckREDC( n, n_, tmp2_2, tmp_2, t_2);

	if ( (d0_s1_1 != d0_s1_2 && d0_s2_1 == d0_s2_2) || (d0_s1_1 == d0_s1_2 && d0_s2_1 != d0_s2_2) ){ //diverge for bit 0 (1 0) or (0 1)
		if ( (d1_s1_1 != d1_s1_2 && d1_s2_1 == d1_s2_2) or (d1_s1_1 == d1_s1_2 && d1_s2_1 != d1_s2_2) ){ //diverge for bit 0, diverge for bit 1 (1 0) or (0 1)
			//printf ("debug3\n");
			return 3;
		} else if ( d1_s1_1 == d1_s1_2 && d1_s2_1 == d1_s2_2 ) { //diverge for bit 0, converge for bit 1 (0 0)
			//printf ("debug4\n");
			return 4;
		} else {
			return 0;
		}
	} else if (d0_s1_1 == d0_s1_2 && d0_s2_1 == d0_s2_2 ){ //converge for bit 0 (0 0)
		if ( (d1_s1_1 != d1_s1_2 && d1_s2_1 == d1_s2_2) || (d1_s1_1 == d1_s1_2 && d1_s2_1 != d1_s2_2) ){ //converge for bit 0, diverge for bit 1 (1 0) or (0 1)
			//printf ("debug1\n");
			return 1;
		} else if ( d1_s1_1 == d1_s1_2 && d1_s2_1 == d1_s2_2 ){ //converge for bit 0, converge for bit 1 (0 0)
			//printf ("debug2\n");
			return 2;
		} else {
			return 0;
		}
	} else {
		return 0;
	}
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	}
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}

	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;

	return time_interval_s + time_interval_ns;
}

//L1 enabled. (-keep -Xptxas -dlcm=ca --optimize 0)  ///enable optimization, enable unroll, add bit length in mpz (only for bit changing ops, others use constant iterations and unroll), use shared memory for everything (meanwhile avoid intialization)?
int main (int argc, char *argv[]) {

	///////input control
	if (argc < 3){
		exit(EXIT_FAILURE);
	}

	long x = strtol(argv[1], NULL, 10);
	long long unsigned pairs = x;
	unsigned thread_num = 2;
	long long unsigned data_num = pairs * thread_num;

	///////host memory
	long long int *clockTable_h;
	clockTable_h = (long long int*) malloc( 4 * sizeof(long long int));

	cuda_mpz_t h_n;
	cuda_mpz_t h_n_;
	cuda_mpz_t h_r2;
//	int rl = 70;

//	cuda_mpz_init(&h_n);
//	cuda_mpz_init(&h_n_);
//	cuda_mpz_init(&h_r2);

	///////get n
	char n_input[] = "00000038f6e8cfba55dd0e47";
	cuda_mpz_set_str_host(&h_n, n_input);

	///////get n_
	char n__input[] = "0000002e8457440e0d93c489";
	cuda_mpz_set_str_host(&h_n_, n__input);

	///////get r2
	char r2_input[] = "0000003709d17d8f8686609f";
	cuda_mpz_set_str_host(&h_r2, r2_input);

	///////get e
	char e_input[] = "101";
	//char e_input[] = "1011011001001001010011110110010101010111001010110101111000111100001";

	int e_bitsLength = (int)strlen(e_input);
	int* eBits = (int *) malloc(sizeof(int) * e_bitsLength);

	int* eBits_d;
	hipMalloc((void **) &eBits_d, sizeof(int) * e_bitsLength);

	int e_iterator = e_bitsLength - 1;
	while ( e_iterator > 0){
        if( e_input[e_bitsLength - 1 - e_iterator] == '1'){
            eBits[e_iterator] = 1;
        }
        else{
            eBits[e_iterator] = 0;
        }
        e_iterator--;
	}
	eBits[e_iterator] = 1;
	hipMemcpy(eBits_d, eBits, sizeof(int) * e_bitsLength, hipMemcpyHostToDevice);

	///////get d
	//char d_input[] = "1011011001001001010011110110010101010111001010110101111000111100001"; //big endian 67 bits
	char d_input[] = "1000100010110110111110111000110000000001011000001000011010101101000101"; //big endian 70 bits
	//char d_input[] = "101";

	int d_bitsLength = (int)strlen(d_input);

	int* dBits = (int *) malloc(sizeof(int) * d_bitsLength);

	int* dBits_d;
	hipMalloc((void **) &dBits_d, sizeof(int) * d_bitsLength);

	int d_iterator = d_bitsLength - 1;
	while ( d_iterator > 0){
        if( d_input[d_bitsLength - 1 - d_iterator] == '1'){//little endian
            dBits[d_iterator] = 1;
        }
        else{
            dBits[d_iterator] = 0;
        }
        d_iterator--;
	}
	dBits[d_iterator] = 1;

	hipMemcpy(dBits_d, dBits, sizeof(int) * d_bitsLength, hipMemcpyHostToDevice);

	///////device memory
	//unsigned varSize = sizeof(cuda_mpz_t) * thread_num;

	long long int *clockTable_d;
//	cuda_mpz_t *tmp;
//	cuda_mpz_t *tmp2;
//	cuda_mpz_t *d_t;
//	cuda_mpz_t *_x1_cuda_mpz;
//	cuda_mpz_t *_x2_cuda_mpz;
	hipMalloc((void **) &clockTable_d, 4 * sizeof(long long int));
//	hipMalloc((void **) &tmp, varSize);
//	hipMalloc((void **) &tmp2, varSize);
//	hipMalloc((void **) &d_t, varSize);
//	hipMalloc((void **) &_x2_cuda_mpz, varSize);
//	hipMalloc((void **) &_x1_cuda_mpz, varSize);

	////////////////////////////////////////////////////////////////initialize
	//init<<<1, thread_num>>>(_x1_cuda_mpz, _x2_cuda_mpz, tmp, tmp2, d_t);
	//hipDeviceSynchronize();

	///////get Messages
	long long unsigned mesSize = sizeof(cuda_mpz_t) * data_num;
	cuda_mpz_t *myMes1_h;
	myMes1_h = (cuda_mpz_t*) malloc (mesSize * 4); //CPU list converge for bit 0, diverge for bit 1
	//cuda_mpz_t *myMes2_h;
	//myMes2_h = (cuda_mpz_t*) malloc (mesSize); //CPU list converge for bit 0 and converge for bit 1
	//cuda_mpz_t *myMes3_h;
	//myMes3_h = (cuda_mpz_t*) malloc (mesSize); //CPU list diverge for bit 0 and converge for bit 1

//	for(long long unsigned i = 0; i < data_num * 4; i++){
//		cuda_mpz_init(&myMes1_h[i]);
//		//cuda_mpz_init(&myMes2_h[i]);
//		//cuda_mpz_init(&myMes3_h[i]);
//	}

	cuda_mpz_t *myMes1_d;
	hipMalloc((cuda_mpz_t **) &myMes1_d, mesSize * 4); //GPU

	///////gen_pairs variables
	int	bit1_div_num = 0;
	int nondiv_num = 0;
	int bothdiv_num = 0;
	int	bit0_div_num = 0;

	cuda_mpz_t r1, r2;
	cuda_mpz_t _x1_1, _x1_2, _x2_1, _x2_2;
	cuda_mpz_t _x1_1_temp, _x1_2_temp, _x2_1_temp, _x2_2_temp;
	cuda_mpz_t tmp_1, tmp_2, tmp2_1, tmp2_2, t_1, t_2;

//	cuda_mpz_init(&r1);
//	cuda_mpz_init(&r2);
//	cuda_mpz_init(&_x1_1);
//	cuda_mpz_init(&_x1_2);
//	cuda_mpz_init(&_x2_1);
//	cuda_mpz_init(&_x2_2);
//	cuda_mpz_init(&_x1_1_temp);
//	cuda_mpz_init(&_x1_2_temp);
//	cuda_mpz_init(&_x2_1_temp);
//	cuda_mpz_init(&_x2_2_temp);
//	cuda_mpz_init(&tmp_1);
//	cuda_mpz_init(&tmp_2);
//	cuda_mpz_init(&tmp2_1);
//	cuda_mpz_init(&tmp2_2);
//	cuda_mpz_init(&t_1);
//	cuda_mpz_init(&t_2);

	long check_pre = strtol(argv[2], NULL, 10);
	int known_bits[2048];
	known_bits[0] = 1;//first bit is always 1
	//known_bits[1] = 0;
	//known_bits[2] = 1;
	int known_bits_length = 1;
	//int total_bits_length = 3;
	int div_con = 0;

	///////gmp init
	mpz_t mod;
	mpz_t rand_num;
	mpz_init (mod);
	mpz_init (rand_num);

	mpz_set_str (mod, n_input, 16);

	///////RNG init
	gmp_randstate_t rand_state;
	//gmp_randinit_default (rand_state);
	gmp_randinit_mt(rand_state);
	gmp_randseed_ui (rand_state, time(NULL));
	//gmp_randseed_ui (rand_state, 0);

	//printf("debug1\n");

	printf("current bits: ");
	for(int i = 0; i < known_bits_length; i++){
		printf("%d", known_bits[i]);
	}
	printf("\n");

	while(known_bits_length < d_bitsLength - 1){

		bit1_div_num = 0;
		nondiv_num = 0;
		bothdiv_num = 0;
		bit0_div_num = 0;

		while(1){

			mpz_urandomm (rand_num, rand_state, mod);
			cuda_mpz_set_gmp(&r1, rand_num);
			mpz_urandomm (rand_num, rand_state, mod);
			cuda_mpz_set_gmp(&r2, rand_num);

			div_con = CheckDivExp(&r1, &r2, known_bits, known_bits_length, &_x1_1, &_x1_2, &_x2_1, &_x2_2,
											&_x1_1_temp, &_x1_2_temp, &_x2_1_temp, &_x2_2_temp,
											&tmp_1, &tmp_2, &tmp2_1, &tmp2_2,  &h_r2, &h_n, &h_n_,  &t_1, &t_2, check_pre);

			if (div_con == 1 && bit1_div_num < data_num){
				cuda_mpz_set( &myMes1_h[bit1_div_num], &r1);
				bit1_div_num++;
				cuda_mpz_set( &myMes1_h[bit1_div_num], &r2);
				bit1_div_num++;
			}
			if (div_con == 2 && nondiv_num < data_num){
				cuda_mpz_set( &myMes1_h[nondiv_num + data_num], &r1);
				nondiv_num++;
				cuda_mpz_set( &myMes1_h[nondiv_num + data_num], &r2);
				nondiv_num++;
			}
			if (div_con == 3 && bothdiv_num < data_num){
				cuda_mpz_set( &myMes1_h[bothdiv_num + data_num * 2], &r1);
				bothdiv_num++;
				cuda_mpz_set( &myMes1_h[bothdiv_num + data_num * 2], &r2);
				bothdiv_num++;
			}
			if (div_con == 4 && bit0_div_num < data_num){
				cuda_mpz_set( &myMes1_h[bit0_div_num + data_num * 3], &r1);
				bit0_div_num++;
				cuda_mpz_set( &myMes1_h[bit0_div_num + data_num * 3], &r2);
				bit0_div_num++;
			}
			if (bit1_div_num == data_num && nondiv_num == data_num && bothdiv_num == data_num && bit0_div_num == data_num){
				break;
			}
		}

		long long int sum1 = 0;
		long long int sum2 = 0;
		long long int sum3 = 0;
		long long int sum4 = 0;

		////////////////////////////////////////////////////////////////converge for bit 0, diverge for bit 1
		hipMemcpy(myMes1_d, myMes1_h, mesSize * 4 , hipMemcpyHostToDevice);

		struct timespec ts1;/////////////////////////////////time
		clock_gettime(CLOCK_REALTIME, &ts1);/////////////////////////////////time

		MontSQMLadder<<<1, thread_num>>>(myMes1_d, pairs, h_r2, h_n, h_n_, dBits_d, d_bitsLength, clockTable_d);/////////////////////////////////////////kernel
		hipDeviceSynchronize();

		struct timespec ts2;/////////////////////////////////time
		clock_gettime(CLOCK_REALTIME, &ts2);/////////////////////////////////time
		printf("%llu\n", time_diff(ts1, ts2));/////////////////////////////////time

		hipMemcpy(clockTable_h, clockTable_d, 4 * sizeof(long long int), hipMemcpyDeviceToHost);

		sum1 = clockTable_h[0];
		sum1 = sum1 / pairs;
		sum2 = clockTable_h[1] - clockTable_h[0];
		sum2 = sum2 / pairs;
		sum3 = clockTable_h[2] - clockTable_h[1];
		sum3 = sum3 / pairs;
		sum4 = clockTable_h[3] - clockTable_h[2];
		sum4 = sum4 / pairs;

		long long int diff1 = abs(sum1 - sum2);
		long long int diff2 = abs(sum2 - sum4);
		long long int diff3 = sum1 - sum4;

		printf("%lld %lld %lld %lld %lld %lld %lld %f %f\n", sum1, sum2, sum3, sum4, diff1, diff2, diff3, ((double) diff1) / diff2, ((double) diff2) / diff1);

		if(diff3 > 2000){//bit is 1
			known_bits[known_bits_length] = 1;
			printf("bit is 1.\n");
		}else if(diff3 < -2000){//bit is 0

			known_bits[known_bits_length] = 0;
			printf("bit is 0.\n");
		}else{//EOB
			//printf("end of bits.\n");
			printf("bit not accepted.\n");
			continue;
		}

		known_bits_length++;

		printf("current bits: ");
		for(int i = 0; i < known_bits_length; i++){
			printf("%d", known_bits[i]);
		}
		printf("\n");

		break;
	}

//	known_bits[known_bits_length] = 1;//last bit is always 1
//	printf("bit is 1.\n");
//
//	known_bits_length++;
//
//	printf("current bits: ");
//	for(int i = 0; i < known_bits_length; i++){
//		printf("%d", known_bits[i]);
//	}
//	printf("\n");

	///////gmp clear
	gmp_randclear (rand_state);
	mpz_clear (rand_num);
	mpz_clear (mod);

	////////free device
	hipFree(clockTable_d);
	hipFree(eBits_d);
	hipFree(dBits_d);
	hipFree(myMes1_d);
//	hipFree(tmp);
//	hipFree(tmp2);
//	hipFree(d_t);
//	hipFree(_x1_cuda_mpz);
//	hipFree(_x2_cuda_mpz);

	////////free host
	free(clockTable_h);
	free(myMes1_h);
	//free(myMes2_h);
	//free(myMes3_h);
	free(eBits);
	free(dBits);

    return 0;
}

