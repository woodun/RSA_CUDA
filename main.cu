
/******************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.cu"
#include <time.h>
#include "mpz.h"

//L1 disabled. (nvcc -Xptxas -dlcm=cg --optimize 0 main.cu -o main)
int main (int argc, char *argv[]) {//./main nodiv.txt 1branchcombo0000.txt 2branchcombo0000.txt
	//./main div.txt 1branchcombo0000.txt 3branchcombo0100.txt

	///////input control
	unsigned pairs = 1001;
	long long unsigned samples = pairs * pairs;
	unsigned thread_num = 2;

	///////host memory
	long long int *clockTable_h;
	clockTable_h = (long long int*) malloc(samples * sizeof(long long int));

	mpz_t h_n;
	mpz_t h_n_;
	mpz_t h_r2;
	int rl = 70;

	mpz_init(&h_n);
	mpz_init(&h_n_);
	mpz_init(&h_r2);

	///////get n
	char n_input[] = "00000038f6e8cfba55dd0e47";
	mpz_set_str_host(&h_n, n_input);
	
	///////get n_
	char n__input[] = "0000002e8457440e0d93c489";
	mpz_set_str_host(&h_n_, n__input);

	///////get r2
	char r2_input[] = "0000003709d17d8f8686609f";
	mpz_set_str_host(&h_r2, r2_input);

	///////get Messages
	unsigned mesSize = sizeof(mpz_t) * pairs;
	mpz_t *myMes1_h;
	myMes1_h = (mpz_t*) malloc (mesSize);
	mpz_t *myMes2_h;
	myMes2_h = (mpz_t*) malloc (mesSize);

	for(int i = 0; i < pairs; i++){
		mpz_init(&myMes1_h[i]);
		mpz_init(&myMes2_h[i]);
	}

	///////get Message1
	char* line = NULL;
	size_t len = 0;
	//char test_str[1024];

	FILE* fp2 = fopen(argv[2], "r");//input from pair storage
	if (fp2 == NULL)
	    exit(EXIT_FAILURE);

	int line_num = 0;
	while ((getline(&line, &len, fp2)) != -1) {
		mpz_set_str_host(&myMes1_h[line_num], line);
		//printf("%s\n", mpz_get_str(&test, test_str, 1024));
		line_num++;
		if(line_num == pairs){
			break;
		}
	}
	fclose(fp2);

	///////get Message2
	FILE* fp3 = fopen(argv[3], "r");//input from pair storage
	if (fp3 == NULL)
	    exit(EXIT_FAILURE);

	line_num = 0;
	while ((getline(&line, &len, fp3)) != -1) {
		mpz_set_str_host(&myMes2_h[line_num], line);
		//printf("%s\n", mpz_get_str(&test, test_str, 1024));
		line_num++;
		if(line_num == pairs){
			break;
		}
	}
	fclose(fp3);

	if (line)
	    free(line);

//	///////get Message1
//	char mes1_input[] = "00000000000123456789";
//	mpz_set_str_host(&myMes1_h[0], mes1_input); //input from string
//
//	///////get Message2
//	char mes2_input[] = "00000000000987654321"; //input from string
//	mpz_set_str_host(&myMes2_h[1], mes2_input);

//	//debug
//	char test_str[1024];
//	printf("%s\n", mpz_get_str(&h_n, test_str, 1024));
//	printf("%s\n", mpz_get_str(&h_n_, test_str, 1024));
//	printf("%s\n", mpz_get_str(&h_r2, test_str, 1024));

	mpz_t *myMes1_d;
	hipMalloc((mpz_t **) &myMes1_d, mesSize);
	hipMemcpy(myMes1_d, myMes1_h, mesSize, hipMemcpyHostToDevice);
	mpz_t *myMes2_d;
	hipMalloc((mpz_t **) &myMes2_d, mesSize);
	hipMemcpy(myMes2_d, myMes2_h, mesSize, hipMemcpyHostToDevice);

	///////get e
	char e_input[] = "101";
	int e_bitsLength = (int)strlen(e_input);
	int* eBits = (int *) malloc(sizeof(int) * e_bitsLength);

	int* eBits_d;
	hipMalloc((void **) &eBits_d, sizeof(int) * e_bitsLength);

	int e_iterator = e_bitsLength - 1;
	while ( e_iterator > 0){
        if( e_input[e_bitsLength - 1 - e_iterator] == '1'){
            eBits[e_iterator] = 1;
        }
        else{
            eBits[e_iterator] = 0;
        }
        e_iterator--;
	}
	eBits[e_iterator] = 1;
	hipMemcpy(eBits_d, eBits, sizeof(int) * e_bitsLength, hipMemcpyHostToDevice);

	///////get d
	char d_input[] = "1011011001001001010011110110010101010111001010110101111000111100001";

	int d_bitsLength = (int)strlen(d_input);

	int* dBits = (int *) malloc(sizeof(int) * d_bitsLength);

	int* dBits_d;
	hipMalloc((void **) &dBits_d, sizeof(int) * d_bitsLength);

	int d_iterator = d_bitsLength - 1;
	while ( d_iterator > 0){
        if( d_input[d_bitsLength - 1 - d_iterator] == '1'){
            dBits[d_iterator] = 1;
        }
        else{
            dBits[d_iterator] = 0;
        }
        d_iterator--;
	}
	dBits[d_iterator] = 1;
	hipMemcpy(dBits_d, dBits, sizeof(int) * d_bitsLength, hipMemcpyHostToDevice);

	///////device memory
	unsigned varSize = sizeof(mpz_t) * thread_num;

	long long int *clockTable_d;
	mpz_t *tmp;
	mpz_t *tmp2;
	mpz_t *d_t;
	mpz_t *_x1_mpz;
	mpz_t *_x2_mpz;
	hipMalloc((void **) &clockTable_d, samples * sizeof(long long int));
	hipMalloc((void **) &tmp, varSize);
	hipMalloc((void **) &tmp2, varSize);
	hipMalloc((void **) &d_t, varSize);
	hipMalloc((void **) &_x2_mpz, varSize);
	hipMalloc((void **) &_x1_mpz, varSize);

	init<<<1, thread_num>>>(_x1_mpz, _x2_mpz, tmp, tmp2, d_t);
	hipDeviceSynchronize();

//	printf("x1: %s\n", mpz_get_str(&myMes1_h[0], test_str, 1024));
//	printf("x2: %s\n", mpz_get_str(&myMes1_h[1], test_str, 1024));
//
//	MontSQMLadder<<<1, thread_num>>>(myMes1_d, myMes2_d, pairs, _x1_mpz, _x2_mpz, tmp, tmp2, rl, h_r2, h_n, h_n_, eBits_d, e_bitsLength, clockTable_d, d_t);/////////////////////////////////////////kernel
//	hipDeviceSynchronize();
//
//	hipMemcpy(myMes1_d, _x1_mpz, mesSize, hipMemcpyDeviceToDevice);
//	hipMemcpy(myMes1_h, _x1_mpz, mesSize, hipMemcpyDeviceToHost);
//
//	printf("x1: %s\n", mpz_get_str(&myMes1_h[0], test_str, 1024));
//	printf("x2: %s\n", mpz_get_str(&myMes1_h[1], test_str, 1024));

	MontSQMLadder<<<1, thread_num>>>(myMes1_d, myMes2_d, pairs, _x1_mpz, _x2_mpz, tmp, tmp2, rl, h_r2, h_n, h_n_, dBits_d, d_bitsLength, clockTable_d, d_t);/////////////////////////////////////////kernel
	hipDeviceSynchronize();

//	hipMemcpy(myMes1_h, _x1_mpz, mesSize, hipMemcpyDeviceToHost);
//
//	printf("x1: %s\n", mpz_get_str(&myMes1_h[0], test_str, 1024));
//	printf("x2: %s\n", mpz_get_str(&myMes1_h[1], test_str, 1024));

	hipMemcpy(clockTable_h, clockTable_d, samples * sizeof(long long int), hipMemcpyDeviceToHost);

	FILE *fp1= fopen(argv[1], "w");
	for (int q = 0; q < samples; q++){
		fprintf(fp1, "%lld\n", clockTable_h[q]);
	}
	fclose(fp1);

	////////free device
	hipFree(clockTable_d);
	hipFree(eBits_d);
	hipFree(dBits_d);
	hipFree(myMes1_d);
	hipFree(myMes2_d);
	hipFree(tmp);
	hipFree(tmp2);
	hipFree(d_t);
	hipFree(_x1_mpz);
	hipFree(_x2_mpz);

	////////free host
	free(clockTable_h);
	free(myMes1_h);
	free(myMes2_h);
	free(eBits);
	free(dBits);

    return 0;
}

