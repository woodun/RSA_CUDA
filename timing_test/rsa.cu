#include "hip/hip_runtime.h"
//#define _GNU_SOURCE
#include <stdio.h>
#include <stdint.h>
#include "include/mpz.h"
#include <assert.h>
#include <unistd.h>
//static char usage[] = "usage: %s -n n_samles -t n_div_thread_limit\n";
#define N 32

// find R
// __device__ __host__ int find_r(mpz * r, mpz * n){
//     for(int i = DIGITS_CAPACITY * LOG2_DIGIT_BASE; i>=0; i--){

//     }
// }

// __global__
// void rsa(uint64_t* ms, int n, long long int* t) {
//     long long int t1, t2;
//     uint64_t i;
//     //uint64_t temp;
//     t[threadIdx.x] = threadIdx.x;
//     a[threadIdx.x] = 0;
//     for(i=0;i<10000000;i++){
//         a[threadIdx.x] += 88;
//         if(n==3333333){ //never true, to make sure line above is executed
//             a[threadIdx.x] = 11111;//powf(float(a[threadIdx.x]),float(n));
//         }
//     }
//     t1 = clock64();
//     //temp = powf(float(a[threadIdx.x]),float(n));
//     if(threadIdx.x >= n){ // branch that causes divergence
//         // for(uint64_t j=0;j<10000;j++){
//             //a[threadIdx.x] += powf(float(a[threadIdx.x]),float(n));
            
//         //asm("sub.s64 %rd20, %rd19, 77777;")
//         t[threadIdx.x] += 22;

//         // }
//         //t[threadIdx.x] = 99;
//         //for(j=0;j<77777;j++){
//             // if(n==5555555){ //never true, to make sure line above is executed
//             //     a[threadIdx.x] = 6666;//powf(float(a[threadIdx.x]),float(n));
//             // }
//         //}
//         //a[threadIdx.x] -= temp - 777;
//     } else {
//         //t[threadIdx.x] *= 33;
//         //asm("add.s64 %rd32, %rd33, %rd34;");
//         //a[threadIdx.x] += powf(float(a[threadIdx.x]),float(n));
//         //asm("add.s64 %rd20, %rd19, 77777;");
//     }
//     t2 = clock64();
//     a[threadIdx.x] = t2 - t1; 
//     t[threadIdx.x] = threadIdx.x;
// }



int main(int argc, char ** argv) {
    // char c;
    // //int n_samples = -1;
    // int div_threads = -1;
    // while ((c = getopt(argc, argv, "t:n:")) != -1)
    //   switch (c) {
    //   case 'n':
    //     n_samples = atoi(optarg);
    //     break;
    //   case 't':
    //     div_threads = atoi(optarg);
    //     break;
    //   case '?':
    //     fprintf(stderr, usage, argv[0]);
    //     exit(1);
    //     break;
    // }
    mpz_t *a, *b, *t;
    a = (mpz_t*) malloc(sizeof(mpz_t));
    b = (mpz_t*) malloc(sizeof(mpz_t));
    t = (mpz_t*) malloc(sizeof(mpz_t));
    mpz_init(a);
    mpz_init(b);
    mpz_init(t);
    mpz_set_str(a, "12345679");
    mpz_print(a);
    printf("a is zero? (0): %d\n", mpz_is_zero(a));
    mpz_bit_rshift(a,2);
    mpz_print(a);
    mpz_bit_rshift(a,26);
    mpz_print(a);
    mpz_bit_rshift(a,1000000);
    mpz_print(a);
    printf("a is zero? (1): %d\n", mpz_is_zero(a));

    //printf("a is zero? (0): %d\n", mpz_is_zero(a));
    //mpz_set_str(a, "0");
    //printf("a is zero? (1): %d\n", mpz_is_zero(a));
    //printf("a is zero? (0): %d\n", mpz_is_zero(a));
    //mpz_print(a);
    //mpz_bit_rshift(a);
    //printf("a is zero? (1): %d\n", mpz_is_zero(a));


    printf("The end\n");

    // if(n_samples <= 0 || div_threads <= 0){
    //     fprintf(stderr, "Wrong parameters! Must specify -n and -t\n");
    //     exit(1);
    // }

    //allocate
    // uint64_t *probes;
    // probes = (uint64_t*) malloc(n_samples * sizeof(uint64_t));
    // assert(probes);
    // long long int* a, *t;
    // long long int* a_d, *t_d;
    // hipMalloc(&a_d, sizeof(long long int) * N);
    // hipMalloc(&t_d, sizeof(long long int) * N);
    // a = (long long int *) malloc(sizeof(long long int) * N);
    // t = (long long int *) malloc(sizeof(long long int) * N);
    //hipMemset(a_d, 333, N);
    //hipEvent_t start, stop;
    //float time;
    //hipEventCreate(&start);
    //hipEventCreate(&stop);
    
    // for(int i=0;i<n_samples;i++){
    //     //*a = 1;
    //     //hipEventRecord(start, 0);
    //     loop<<<1, N>>>(a_d, 31, t_d);
        
    //     //hipEventRecord(stop, 0);
    //     //hipEventSynchronize(stop);
    //     hipDeviceSynchronize();
    //     hipMemcpy(a, a_d, sizeof(long long int) * N, hipMemcpyDefault);
    //     hipMemcpy(t, t_d, sizeof(long long int) * N, hipMemcpyDefault);

    //     //hipEventElapsedTime(&time, start, stop);
    //     printf ("%lld\n", *a);

    // }

    // printf("---------\n");

    // for(int i=0;i<n_samples;i++){
    //     //*a = 1;
    //     //hipEventRecord(start, 0);
    //     loop<<<1, N>>>(a_d, 100, t_d);
        
    //     //hipEventRecord(stop, 0);
    //     //hipEventSynchronize(stop);
    //     hipDeviceSynchronize();
    //     hipMemcpy(a, a_d, sizeof(long long int) * N, hipMemcpyDefault);
    //     hipMemcpy(t, t_d, sizeof(long long int) * N, hipMemcpyDefault);

    //     //hipEventElapsedTime(&time, start, stop);
    //     printf ("%lld\n", *a);

    // }

    // // // printf("a: %d\n", *a);
    // // // for (int j=0;j<N;j++){
    // // //     printf("%d:%d\t", j, t[j]);
    // // // }
    // // // printf("\n");


    // hipFree(&a);
    // hipFree(&t);
    // free(a);
    // free(t);

    // Retrieve result from device and store it in host array

    return 0;
}
